
#include <hip/hip_runtime.h>
# include <stdio.h>

// CUDA kernel function
__global__ void helloFromGPU(){
    printf("Hello world! GPU working. Thread ID: %d\n", threadIdx.x);
}

int main() {
    printf("This is your CPU speaking....\n");

    // Launching kernel with 1 block and 10 threads
    helloFromGPU<<<1, 10>>>();

    // Wait for GPU to finish before exiting
    hipDeviceSynchronize();

    return 0;
}